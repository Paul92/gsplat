#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "types.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#define NORMALIZE_EPS 1.0E-12F

namespace gsplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Rasterization to Pixels Backward Pass
 ****************************************************************************/

template <uint32_t COLOR_DIM, typename S>
__global__ void rasterize_to_pixels_bwd_radegs_kernel(
    const uint32_t C,
    const uint32_t N,
    const uint32_t n_isects,
    const bool packed,
    // fwd inputs
    const vec2<S> *__restrict__ means2d, // [C, N, 2] or [nnz, 2]
    const vec3<S> *__restrict__ conics,  // [C, N, 3] or [nnz, 3]
    const S *__restrict__ colors,      // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    const S *__restrict__ opacities,   // [C, N] or [nnz]
    const vec2<S> *__restrict__ ray_planes, // [C, N, 2]
    const vec3<S> *__restrict__ normals, // [C, N, 3]
    const S *__restrict__ ts,            // [C, N]
    const S *__restrict__ backgrounds, // [C, COLOR_DIM] or [nnz, COLOR_DIM]
    const bool *__restrict__ masks,    // [C, tile_height, tile_width]
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    const uint32_t tile_width,
    const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    // fwd outputs
    const S *__restrict__ render_alphas,  // [C, image_height, image_width, 1]
    const S *__restrict__ render_depths,  // [C, image_height, image_width, 1]
    const vec3<S> *__restrict__ render_normals,  // [C, image_height, image_width, 3]
    const int32_t *__restrict__ last_ids, // [C, image_height, image_width]
    const int32_t *__restrict__ max_ids, // [C, image_height, image_width]
    // grad outputs
    const S *__restrict__ v_render_colors, // [C, image_height, image_width,
                                           // COLOR_DIM]
    const S *__restrict__ v_render_alphas, // [C, image_height, image_width, 1]
    const S *__restrict__ v_render_depths, // [C, image_height, image_width, 1]
    const S *__restrict__ v_render_mdepths, // [C, image_height, image_width, 1]
    const S *__restrict__ v_render_normals, // [C, image_height, image_width, 3]
    // grad inputs
    vec2<S> *__restrict__ v_means2d_abs, // [C, N, 2] or [nnz, 2]
    vec2<S> *__restrict__ v_means2d,     // [C, N, 2] or [nnz, 2]
    vec3<S> *__restrict__ v_conics,      // [C, N, 3] or [nnz, 3]
    S *__restrict__ v_colors,   // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    S *__restrict__ v_opacities, // [C, N] or [nnz]
    vec2<S> *__restrict__ v_camera_planes,
    vec2<S> *__restrict__ v_ray_planes,
    vec3<S> *__restrict__ v_normals,
    S *__restrict__ v_ts,
    mat3<S> *__restrict__ K
) {
    auto block = cg::this_thread_block();
    uint32_t camera_id = block.group_index().x;
    uint32_t tile_id =
        block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    tile_offsets += camera_id * tile_height * tile_width;
    render_alphas += camera_id * image_height * image_width;
    last_ids += camera_id * image_height * image_width;
    v_render_colors += camera_id * image_height * image_width * COLOR_DIM;
    v_render_alphas += camera_id * image_height * image_width;
    v_render_depths += camera_id * image_height * image_width;
    v_render_normals += camera_id * image_height * image_width * 3;

    if (backgrounds != nullptr) {
        backgrounds += camera_id * COLOR_DIM;
    }
    if (masks != nullptr) {
        masks += camera_id * tile_height * tile_width;
    }

    // when the mask is provided, do nothing and return if
    // this tile is labeled as False
    if (masks != nullptr && !masks[tile_id]) {
        return;
    }

    const S px = (S)j + 0.5f;
    const S py = (S)i + 0.5f;
    // clamp this value to the last pixel
    const int32_t pix_id =
        min(i * image_width + j, image_width * image_height - 1);

    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    const uint32_t num_batches =
        (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s; // [block_size]
    vec3<S> *xy_opacity_batch =
        reinterpret_cast<vec3<float> *>(&id_batch[block_size]); // [block_size]
    vec3<S> *conic_batch =
        reinterpret_cast<vec3<float> *>(&xy_opacity_batch[block_size]
        );                                         // [block_size]
    S *rgbs_batch = (S *)&conic_batch[block_size]; // [block_size * COLOR_DIM]
    vec2<S> *ray_planes_batch =
        reinterpret_cast<vec2<float> *>(&rgbs_batch[block_size * COLOR_DIM]); // [block_size]
    S *ts_batch = (S *)&ray_planes_batch[block_size]; // [block_size]
    vec3<S> *normals_batch =
        reinterpret_cast<vec3<float> *>(&ts_batch[block_size]); // [block_size]
    vec2<S> *camera_planes_batch =
        reinterpret_cast<vec2<float> *>(&normals_batch[block_size]); // [block_size * 3]


    // this is the T AFTER the last gaussian in this pixel
    S T_final = 1.0f - render_alphas[pix_id];
    S T = T_final;
    // the contribution from gaussians behind the current one
    S buffer[COLOR_DIM] = {0.f};
    // index of last gaussian to contribute to this pixel
    const int32_t bin_final = inside ? last_ids[pix_id] : 0;

    // df/d_out for this pixel
    S v_render_c[COLOR_DIM];
    GSPLAT_PRAGMA_UNROLL
    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
        v_render_c[k] = v_render_colors[pix_id * COLOR_DIM + k];
    }
    const S v_render_a = v_render_alphas[pix_id];


    const float ddelx_dx = 0.5 * image_width;
    const float ddely_dy = 0.5 * image_height;


    float accum_rec = { 0 };
    float dL_dpixel;
    vec3<S> accum_coord_rec{0, 0, 0};
    vec3<S> dL_dpixel_coord{0, 0, 0};
    float accum_t_rec = 0;
    float accum_alpha_rec = 0;
    vec3<S> accum_normal_rec{0, 0, 0};
    vec3<S> dL_dpixel_mcoord;

    float dL_dt;

    float dL_dpixel_t = 0;
    float dL_dpixel_mt = 0;
    float dL_dalpha = 0;

    const float w_final = inside ? render_alphas[pix_id] : 0;
    const vec2<S> pixf = { (S)j, (S)i }; // TODO: check if order is correct
    const vec2<S> pixnf = {(pixf.x-image_width/2.f)/(*K)[0][0],(pixf.y-image_height/2.f)/(*K)[1][1]};
    const float ln = sqrt(pixnf.x*pixnf.x+pixnf.y*pixnf.y+1);
    vec3<S> dL_dpixel_normal = {0.f, 0.f, 0.f};


    float last_alpha = 0;
    float last_color[COLOR_DIM] = { 0 };
    float last_coord[3] = { 0 };
    float last_t = 0;
    float last_dL_dw = 0;
    vec3<S> last_normal{0, 0, 0};


    if (inside) {
        dL_dalpha = v_render_alphas[pix_id];
        float ww = w_final*w_final;

        {
           float dL_dpixel_depth_w = v_render_depths[pix_id];
           float pixel_accum_depth = render_depths[pix_id] * render_alphas[pix_id];
           dL_dalpha -= dL_dpixel_depth_w*pixel_accum_depth/ww;
           dL_dpixel_t = dL_dpixel_depth_w / w_final / ln;
           dL_dpixel_mt = v_render_mdepths[pix_id] / ln;
        }

        {
            vec3<S> dL_dpixel_normaln;
            dL_dpixel_normaln.x = v_render_normals[pix_id * 3];
            dL_dpixel_normaln.y = v_render_normals[pix_id * 3 + 1];
            dL_dpixel_normaln.z = v_render_normals[pix_id * 3 + 2];

            glm::vec3 normaln = glm::vec3(render_normals[pix_id]);
            S normal_len = glm::length(normaln);
            glm::vec3 dL;
            if (normal_len < NORMALIZE_EPS)
                dL = dL_dpixel_normaln / NORMALIZE_EPS;
            else
                dL = (dL_dpixel_normaln - glm::dot(dL_dpixel_normaln, normaln) * normaln) / normal_len;

            dL_dpixel_normal = dL;
        }
    }

    uint32_t contributor = range_end - range_start;
    const int last_contributor = inside ? last_ids[pix_id] : 0;
    const int max_contributor = inside ? max_ids[pix_id] : 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const uint32_t tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int32_t warp_bin_final =
        cg::reduce(warp, bin_final, cg::greater<int>());
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        // These values can be negative so must be int32 instead of uint32
        const int32_t batch_end = range_end - 1 - block_size * b;
        const int32_t batch_size = min(block_size, batch_end + 1 - range_start);
        const int32_t idx = batch_end - tr;
        if (idx >= range_start) {
            int32_t g = flatten_ids[idx]; // flatten index in [C * N] or [nnz]
            id_batch[tr] = g;
            const vec2<S> xy = means2d[g];
            const S opac = opacities[g];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            conic_batch[tr] = conics[g];
            GSPLAT_PRAGMA_UNROLL
            for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                rgbs_batch[tr * COLOR_DIM + k] = colors[g * COLOR_DIM + k];
            }
            ray_planes_batch[tr] = ray_planes[g];
            ts_batch[tr] = ts[g];
            normals_batch[tr] = normals[g];
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (uint32_t t = max(0, batch_end - warp_bin_final); t < batch_size;
             ++t) {
            bool valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            S alpha;
            S opac;
            vec2<S> delta;
            vec3<S> conic;
            S vis;

            if (valid) {
                conic = conic_batch[t];
                vec3<S> xy_opac = xy_opacity_batch[t];
                opac = xy_opac.z;
                delta = {xy_opac.x - px, xy_opac.y - py};
                S sigma = 0.5f * (conic.x * delta.x * delta.x +
                                  conic.z * delta.y * delta.y) +
                          conic.y * delta.x * delta.y;
                vis = __expf(-sigma);
                alpha = min(0.999f, opac * vis);
                if (sigma < 0.f || alpha < 1.f / 255.f) {
                    valid = false;
                }
            }

            // if all threads are inactive in this warp, skip this loop
            if (!warp.any(valid)) {
                continue;
            }
            S v_rgb_local[COLOR_DIM] = {0.f};
            vec3<S> v_conic_local = {0.f, 0.f, 0.f};
            vec2<S> v_xy_local = {0.f, 0.f};
            vec2<S> v_xy_abs_local = {0.f, 0.f};
            S v_opacity_local = 0.f;
            vec2<S> v_ray_plane_local = {0.f, 0.f};
            vec3<S> v_normal_local = {0.f, 0.f, 0.f};
            S v_ts_local = 0.f;
            vec2<S> ray_plane;
            // initialize everything to 0, only set if the lane is valid
            if (valid) {
                // compute the current T for this gaussian
                S ra = 1.0f / (1.0f - alpha);
                T *= ra;
                // update v_rgb for this gaussian
                const S fac = alpha * T;
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_rgb_local[k] = fac * v_render_c[k];
                }
                // contribution from this pixel
                S v_alpha = 0.f;
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    v_alpha +=
                        (rgbs_batch[t * COLOR_DIM + k] * T - buffer[k] * ra) *
                        v_render_c[k];
                }

                v_alpha += T_final * ra * v_render_a;
                // contribution from background pixel
                if (backgrounds != nullptr) {
                    S accum = 0.f;
                    GSPLAT_PRAGMA_UNROLL
                    for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                        accum += backgrounds[k] * v_render_c[k];
                    }
                    v_alpha += -T_final * ra * accum;
                }

                if (opac * vis <= 0.999f) {
                    const S v_sigma = -opac * vis * v_alpha;
                    v_conic_local = {
                        0.5f * v_sigma * delta.x * delta.x,
                        v_sigma * delta.x * delta.y,
                        0.5f * v_sigma * delta.y * delta.y
                    };
                    v_xy_local = {
                        v_sigma * (conic.x * delta.x + conic.y * delta.y),
                        v_sigma * (conic.y * delta.x + conic.z * delta.y)
                    };
                    if (v_means2d_abs != nullptr) {
                        v_xy_abs_local = {abs(v_xy_local.x), abs(v_xy_local.y)};
                    }
                    v_opacity_local = v_alpha;
                }

                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    buffer[k] += rgbs_batch[t * COLOR_DIM + k] * fac;
                }

                { // Depth
                    const float t_center = ts_batch[t];
                    ray_plane = ray_planes_batch[t];
                    float tt = t_center + (ray_plane.x * delta.x + ray_plane.y * delta.y);
                    accum_t_rec = last_alpha * last_t + (1.f - last_alpha) * accum_t_rec;
                    last_t = tt;
                    v_opacity_local += (t - accum_t_rec) * dL_dpixel_t;
                    dL_dt = fac * dL_dpixel_t;
                    if (contributor == batch_end - max_contributor-1) {
                        dL_dt += dL_dpixel_mt;
                    }
                }

                { // Normal
                    v_ts_local = dL_dt;
                    v_ray_plane_local = {dL_dt * delta.x / (*K)[0][0], dL_dt * delta.y / (*K)[1][1]};

                    vec3<S> normal = normals_batch[t];
                    // Update last color (to be used in the next iteration)
                    accum_normal_rec = last_alpha * last_normal + (1.f - last_alpha) * accum_normal_rec;
                    last_normal = normal;
                    vec3<S> normal_contrib = (normal - accum_normal_rec) * dL_dpixel_normal;
                    v_opacity_local += normal_contrib.x + normal_contrib.y + normal_contrib.z;

                    v_normal_local = fac * dL_dpixel_normal;
                }


            }
            warpSum<COLOR_DIM, S>(v_rgb_local, warp);
            warpSum<decltype(warp), S>(v_conic_local, warp);
            warpSum<decltype(warp), S>(v_xy_local, warp);
            if (v_means2d_abs != nullptr) {
                warpSum<decltype(warp), S>(v_xy_abs_local, warp);
            }
            warpSum<decltype(warp), S>(v_opacity_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t]; // flatten index in [C * N] or [nnz]
                S *v_rgb_ptr = (S *)(v_colors) + COLOR_DIM * g;
                GSPLAT_PRAGMA_UNROLL
                for (uint32_t k = 0; k < COLOR_DIM; ++k) {
                    gpuAtomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                S *v_conic_ptr = (S *)(v_conics) + 3 * g;
                gpuAtomicAdd(v_conic_ptr, v_conic_local.x);
                gpuAtomicAdd(v_conic_ptr + 1, v_conic_local.y);
                gpuAtomicAdd(v_conic_ptr + 2, v_conic_local.z);

                S *v_xy_ptr = (S *)(v_means2d) + 2 * g;
                gpuAtomicAdd(v_xy_ptr, v_xy_local.x);
                gpuAtomicAdd(v_xy_ptr + 1, v_xy_local.y);

                if (v_means2d_abs != nullptr) {
                    S *v_xy_abs_ptr = (S *)(v_means2d_abs) + 2 * g;
                    gpuAtomicAdd(v_xy_abs_ptr, v_xy_abs_local.x);
                    gpuAtomicAdd(v_xy_abs_ptr + 1, v_xy_abs_local.y);
                }

                gpuAtomicAdd(v_opacities + g, vis * v_opacity_local);

                gpuAtomicAdd(v_ts + g, v_ts_local);

                S *v_ray_planes_ptr = (S *)(v_ray_planes) + 2 * g;
                gpuAtomicAdd(v_ray_planes_ptr, v_ray_plane_local.x);
                gpuAtomicAdd(v_ray_planes_ptr+1, v_ray_plane_local.y);

                S *v_normals_ptr = (S *)(v_normals) + 3 * g;
                gpuAtomicAdd(v_normals_ptr, v_normal_local.x);
                gpuAtomicAdd(v_normals_ptr + 1, v_normal_local.y);
                gpuAtomicAdd(v_normals_ptr + 2, v_normal_local.z);
            }
        }
    }
}

template <uint32_t CDIM>
std::tuple<
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor>
call_kernel_with_dim(
    // Gaussian parameters
    const torch::Tensor &means2d,                   // [C, N, 2] or [nnz, 2]
    const torch::Tensor &conics,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &colors,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &opacities,                 // [C, N] or [nnz]
    const torch::Tensor &camera_planes,
    const torch::Tensor &ray_planes,
    const torch::Tensor &normals,
    const torch::Tensor &ts,
    const torch::Tensor &K,
    const at::optional<torch::Tensor> &backgrounds, // [C, 3]
    const at::optional<torch::Tensor> &masks, // [C, tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_depths, // [C, image_height, image_width, 1]
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &render_normals, // [C, image_height, image_width, 3]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    const torch::Tensor &max_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &v_render_depths, // [C, image_height, image_width, 1]
    const torch::Tensor &v_render_mdepths, // [C, image_height, image_width, 1]
    const torch::Tensor &v_render_normals, // [C, image_height, image_width, 3]
    // options
    bool absgrad
) {
    printf("Called rasterize_to_pixels kernel caller\n");

    GSPLAT_DEVICE_GUARD(means2d);
    GSPLAT_CHECK_INPUT(means2d);
    GSPLAT_CHECK_INPUT(conics);
    GSPLAT_CHECK_INPUT(colors);
    GSPLAT_CHECK_INPUT(opacities);
    GSPLAT_CHECK_INPUT(camera_planes);
    GSPLAT_CHECK_INPUT(ray_planes);
    GSPLAT_CHECK_INPUT(normals);
    GSPLAT_CHECK_INPUT(ts);
    GSPLAT_CHECK_INPUT(K);
    GSPLAT_CHECK_INPUT(tile_offsets);
    GSPLAT_CHECK_INPUT(flatten_ids);
    GSPLAT_CHECK_INPUT(render_depths);
    GSPLAT_CHECK_INPUT(render_alphas);
    GSPLAT_CHECK_INPUT(render_normals);
    GSPLAT_CHECK_INPUT(last_ids);
    GSPLAT_CHECK_INPUT(max_ids);
    GSPLAT_CHECK_INPUT(v_render_colors);
    GSPLAT_CHECK_INPUT(v_render_alphas);
    GSPLAT_CHECK_INPUT(v_render_depths);
    GSPLAT_CHECK_INPUT(v_render_mdepths);
    GSPLAT_CHECK_INPUT(v_render_normals);
    if (backgrounds.has_value()) {
        GSPLAT_CHECK_INPUT(backgrounds.value());
    }
    if (masks.has_value()) {
        GSPLAT_CHECK_INPUT(masks.value());
    }

    bool packed = means2d.dim() == 2;

    uint32_t C = tile_offsets.size(0);         // number of cameras
    uint32_t N = packed ? 0 : means2d.size(1); // number of gaussians
    uint32_t n_isects = flatten_ids.size(0);
    uint32_t COLOR_DIM = colors.size(-1);
    uint32_t tile_height = tile_offsets.size(1);
    uint32_t tile_width = tile_offsets.size(2);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {C, tile_height, tile_width};

    torch::Tensor v_means2d = torch::zeros_like(means2d);
    torch::Tensor v_conics = torch::zeros_like(conics);
    torch::Tensor v_colors = torch::zeros_like(colors);
    torch::Tensor v_opacities = torch::zeros_like(opacities);
    torch::Tensor v_means2d_abs;
    if (absgrad) {
        v_means2d_abs = torch::zeros_like(means2d);
    }
    torch::Tensor v_camera_planes = torch::zeros_like(camera_planes);
    torch::Tensor v_ray_planes = torch::zeros_like(ray_planes);
    torch::Tensor v_normals = torch::zeros_like(normals);
    torch::Tensor v_ts = torch::zeros_like(ts);

    if (n_isects) {
        const uint32_t shared_mem =
            tile_size * tile_size *
            (sizeof(int32_t) + sizeof(vec3<float>) + sizeof(vec3<float>) +
             sizeof(float) * COLOR_DIM + sizeof(vec2<float>) + sizeof(float) + sizeof(vec3<float>));
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

        if (hipFuncSetAttribute(reinterpret_cast<const void*>(
                rasterize_to_pixels_bwd_radegs_kernel<CDIM), float>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                shared_mem
            ) != hipSuccess) {
            AT_ERROR(
                "Failed to set maximum shared memory size (requested ",
                shared_mem,
                " bytes), try lowering tile_size."
            );
        }

        rasterize_to_pixels_bwd_radegs_kernel<CDIM, float>
            <<<blocks, threads, shared_mem, stream>>>(
                C,
                N,
                n_isects,
                packed,
                reinterpret_cast<vec2<float> *>(means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(conics.data_ptr<float>()),
                colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                reinterpret_cast<vec2<float> *>(ray_planes.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(normals.data_ptr<float>()),
                ts.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                masks.has_value() ? masks.value().data_ptr<bool>() : nullptr,
                image_width,
                image_height,
                tile_size,
                tile_width,
                tile_height,
                tile_offsets.data_ptr<int32_t>(),
                flatten_ids.data_ptr<int32_t>(),
                render_depths.data_ptr<float>(),
                render_alphas.data_ptr<float>(),
                reinterpret_cast<vec3<float> *>(render_normals.data_ptr<float>()),
                last_ids.data_ptr<int32_t>(),
                max_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(),
                v_render_alphas.data_ptr<float>(),
                v_render_depths.data_ptr<float>(),
                v_render_mdepths.data_ptr<float>(),
                v_render_normals.data_ptr<float>(),
                absgrad ? reinterpret_cast<vec2<float> *>(
                              v_means2d_abs.data_ptr<float>()
                          )
                        : nullptr,
                reinterpret_cast<vec2<float> *>(v_means2d.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(v_conics.data_ptr<float>()),
                v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>(),
                reinterpret_cast<vec2<float> *>(v_camera_planes.data_ptr<float>()),
                reinterpret_cast<vec2<float> *>(v_ray_planes.data_ptr<float>()),
                reinterpret_cast<vec3<float> *>(v_normals.data_ptr<float>()),
                v_ts.data_ptr<float>(),
                reinterpret_cast<mat3<float> *>(K.data_ptr<float>())
            );
    }

    return std::make_tuple(
        v_means2d_abs, v_means2d, v_conics, v_colors, v_opacities,
        v_camera_planes, v_ray_planes, v_normals, v_ts
    );
}

std::tuple<
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor,
    torch::Tensor>
rasterize_to_pixels_bwd_radegs_tensor(
    // Gaussian parameters
    const torch::Tensor &means2d,                   // [C, N, 2] or [nnz, 2]
    const torch::Tensor &conics,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &colors,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &opacities,                 // [C, N] or [nnz]
    const torch::Tensor &camera_planes,
    const torch::Tensor &ray_planes,
    const torch::Tensor &normals,
    const torch::Tensor &ts,
    const torch::Tensor &K,
    const at::optional<torch::Tensor> &backgrounds, // [C, 3]
    const at::optional<torch::Tensor> &masks, // [C, tile_height, tile_width]
    // image size
    const uint32_t image_width,
    const uint32_t image_height,
    const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // forward outputs
    const torch::Tensor &render_depths, // [C, image_height, image_width, 1]
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &render_normals, // [C, image_height, image_width, 3]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    const torch::Tensor &max_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &v_render_depths, // [C, image_height, image_width, 1]
    const torch::Tensor &v_render_mdepths, // [C, image_height, image_width, 1]
    const torch::Tensor &v_render_normals, // [C, image_height, image_width, 3]
    // options
    bool absgrad
) {

    GSPLAT_CHECK_INPUT(colors);
    uint32_t COLOR_DIM = colors.size(-1);

#define __GS__CALL_(N)                                                         \
    case N:                                                                    \
        return call_kernel_with_dim<N>(                                        \
            means2d,                                                           \
            conics,                                                            \
            colors,                                                            \
            opacities,                                                         \
            camera_planes,                                                     \
            ray_planes,                                                        \
            normals,                                                           \
            ts,                                                                \
            K,                                                                 \
            backgrounds,                                                       \
            masks,                                                             \
            image_width,                                                       \
            image_height,                                                      \
            tile_size,                                                         \
            tile_offsets,                                                      \
            flatten_ids,                                                       \
            render_depths,                                                     \
            render_alphas,                                                     \
            render_normals,                                                    \
            last_ids,                                                          \
            max_ids,                                                          \
            v_render_colors,                                                   \
            v_render_alphas,                                                   \
            v_render_depths,                                                   \
            v_render_mdepths,                                                   \
            v_render_normals,                                                  \
            absgrad                                                            \
        );

    switch (COLOR_DIM) {
        __GS__CALL_(1)
        __GS__CALL_(2)
        __GS__CALL_(3)
        __GS__CALL_(4)
        __GS__CALL_(5)
        __GS__CALL_(8)
        __GS__CALL_(9)
        __GS__CALL_(16)
        __GS__CALL_(17)
        __GS__CALL_(32)
        __GS__CALL_(33)
        __GS__CALL_(64)
        __GS__CALL_(65)
        __GS__CALL_(128)
        __GS__CALL_(129)
        __GS__CALL_(256)
        __GS__CALL_(257)
        __GS__CALL_(512)
        __GS__CALL_(513)
    default:
        AT_ERROR("Unsupported number of channels: ", COLOR_DIM);
    }
}

} // namespace gsplat
